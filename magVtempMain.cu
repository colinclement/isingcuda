#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <float.h>
#include <string.h>

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hiprand.h>
#include <hip/hip_runtime_api.h> //In samples/common/inc
#include <thrust/reduce.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

//#define DBUG //Save stuff to files
#define MOD(x, N) (((x < 0) ? ((x % N) + N) : x) % N)
#ifndef MIN
#define MIN(a, b) ((a > b) ? b : a)
#endif
#ifndef MAX
#define MAX(a, b) ((a > b) ? a : b)
#endif

#define THREADS_PER 8

__global__
void isingSample(int *d_spins, float *d_random, const float T,
                 const int L);

__device__
void chessBoardSample(int *d_spins, float *d_random, const float T,
                      const int L, const int irow, const int icol);

int main(int argc, char **argv){

    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    if (deviceCount == 0){
        fprintf(stderr, "Error: no CUDA supporting devices.\n");
	exit(EXIT_FAILURE);
    }
    int dev = 0; 
    hipSetDevice(dev);
    
    const char *printMSG = "Incorrect number of arguments: Usage: \n\
			    ./cuising filename L Tmin Tmax numTs N_steps period burnin \n";
    if (argc < 9){
        printf("%s", printMSG);
	return 0;
    }
    else if (argc > 9){
        printf("%s", printMSG);
        return 0;
    }

    char *filename = argv[1];
    int L = atoi(argv[2]);
    float Tmin = atof(argv[3]), Tmax = atof(argv[4]);
    int numTs = atoi(argv[5]);
    int N_steps = atoi(argc[6]);
    int period = atoi(argv[7]), burnin = atoi(argv[8]);
    printf("Saving to %s with L=%d, T=%f, every %d steps,\n with burnin=%d\n",
           filename, L, T, period, burnin);

    int N = L*L;

    hiprandGenerator_t rng;
    checkCudaErrors(hiprandCreateGenerator(&rng, HIPRAND_RNG_PSEUDO_DEFAULT));
    checkCudaErrors(hiprandSetPseudoRandomGeneratorSeed(rng, 920989ULL));

    //hipblasHandle_t handle;
    //checkCudaErrors(hipblasCreate(&handle));

    int *h_spins = (int *)malloc(sizeof(int) * N);
    memset(h_spins, 1, sizeof(int) * N);

    for (int i = 0; i < N; i++){
        float r = (float)rand()/RAND_MAX;
        h_spins[i] = (r > 0.5) ? 1 : -1;
    }
    int *d_spins;
    float *d_random;
    checkCudaErrors(hipMalloc((void **)&d_spins, sizeof(int) * N));
    checkCudaErrors(hipMalloc((void **)&d_random, sizeof(float) * N));
    checkCudaErrors(hipMemcpy(d_spins, h_spins, sizeof(int) * N, hipMemcpyHostToDevice));
    checkCudaErrors(hipDeviceSynchronize());

    checkCudaErrors(hiprandGenerateUniform(rng, d_random, N));
    checkCudaErrors(hipDeviceSynchronize());
    float *h_random = (float *)malloc(sizeof(float) * N);
    checkCudaErrors(hipMemcpy(h_random, d_random, sizeof(int) * N, hipMemcpyDeviceToHost));
    checkCudaErrors(hipDeviceSynchronize());

    FILE *fp = fopen("dbug.dat", "w");
    for (int i=0; i < N; i++){
        if (i%L ==0)
            fprintf(fp, "\n");
        fprintf(fp, "%d\t", h_spins[i]);
    }
    for (int i=0; i < N; i++){
        if (i%L ==0)
            fprintf(fp, "\n");
        fprintf(fp, "%f\t", h_random[i]);
    }

    FILE *fpMag = fopen(filename, "w");

    int NUMBLOCKS = ceil((float)L/(float)THREADS_PER);
    int BLOCKMEM = sizeof(int) * (THREADS_PER+2) * (THREADS_PER+2);
    dim3 blocks(NUMBLOCKS, NUMBLOCKS);
    dim3 threads(THREADS_PER, THREADS_PER);

    hipEvent_t start, stop;
    float time = 0.f;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));
    checkCudaErrors(hipEventRecord(start, 0));
   
    float Tstep = (Tmax - Tmin) / ((float) numTs);

    for (int iT = 0; iT < numTs; iT++){
        float temp = Tmax - Tstep * iT;
        float mag = 0.f;
        for (int t = 0; t < burnin; t++){
            isingSample<<<blocks, threads, 
                          BLOCKMEM>>>(d_spins, d_random, temp, L);
            checkCudaErrors(hiprandGenerateUniform(rng, d_random, N));
            checkCudaErrors(hipDeviceSynchronize());
        } 
        
        for (int t = 0; t < N_steps; t++){
            isingSample<<<blocks, threads, 
                          BLOCKMEM>>>(d_spins, d_random, temp, L);
            checkCudaErrors(hiprandGenerateUniform(rng, d_random, N));
            checkCudaErrors(hipDeviceSynchronize());
            if (t % period == 0){
                thrust::device_ptr<int> spinPtr = thrust::device_pointer_cast(d_spins);
                mag += ((float) thrust::reduce(spinPtr, spinPtr + N))/((float) N);
            }
        }
        fprintf(fpMag, "%f\t%f\n", mag/((float)N_steps/period), temp);
    }
    fclose(fpMag);

    checkCudaErrors(hipEventRecord(stop, 0));
    checkCudaErrors(hipEventSynchronize(stop));
    hipEventElapsedTime(&time, start, stop);
    printf("Elapsed time: %f ms, %f ms/step\n", time, time/(float)burnin);
    
    checkCudaErrors(hipMemcpy(h_spins, d_spins, sizeof(int) * N, hipMemcpyDeviceToHost));

    for (int i=0; i < N; i++){
        if (i%L ==0)
            fprintf(fp, "\n");
        fprintf(fp, "%d\t", h_spins[i]);
    }

    fclose(fp);
    hipFree(d_spins);
    hipFree(d_random);
    free(h_spins);
    free(h_random);
    checkCudaErrors(hipGetLastError());

    return EXIT_SUCCESS;
}

__global__
void isingSample(int *d_spins, float *d_random, const float T,
                 const int L){
    int N = L*L;
    int icol = threadIdx.x + blockIdx.x * blockDim.x;
    int irow = threadIdx.y + blockIdx.y * blockDim.y;
    int site = irow * L + icol;
    if (site >= N || icol >=L || irow >= L)
        return;
    int chess = (icol % 2 + irow % 2)%2;
    //int blockChess = (blockIdx.x%2 + blockIdx.y%2)%2;
    //extern __shared__ int *s_spins[];//(blockDim+2)**2

    if (chess == 0)
        chessBoardSample(d_spins, d_random, T, L, irow, icol);
    __syncthreads();
    if (chess == 1)
        chessBoardSample(d_spins, d_random, T, L, irow, icol);

    return;
}

__device__
void chessBoardSample(int *d_spins, float *d_random, const float T,
                      const int L, const int irow, const int icol){
    int site = irow * L + icol;
    int neighSum = 0, r = site, c = site;
    int spin = d_spins[site];

    for (int i =-1; i < 2; i++){
        for (int j=-1; j < 2; j++){
            if (abs(i) != abs(j)){
                r = MOD(irow + i, L);
                c = MOD(icol + j, L);
                neighSum += d_spins[r * L + c];
            }
        }
    }
    float dE = 2 * spin * neighSum;
    if (exp(- dE/T) > d_random[site])
        d_spins[site] = -1 * spin;
    
    return;
}

