#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <float.h>
#include <string.h>

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hiprand.h>
#include <hip/hip_runtime_api.h> //In samples/common/inc
#include <thrust/reduce.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include <sampling.h>

//#define DBUG //Save stuff to files
#define MOD(x, N) (((x < 0) ? ((x % N) + N) : x) % N)
#ifndef MIN
#define MIN(a, b) ((a > b) ? b : a)
#endif
#ifndef MAX
#define MAX(a, b) ((a > b) ? a : b)
#endif

//I actually use THREAD_X+2, THREAD_Y+2
// (THREAD_X+2)*(THREAD_Y+2) < 1024
#define THREAD_X 30 
#define THREAD_Y 30

int main(int argc, char **argv){

    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    if (deviceCount == 0){
        fprintf(stderr, "Error: no CUDA supporting devices.\n");
	exit(EXIT_FAILURE);
    }
    int dev = 0; 
    hipSetDevice(dev);
    
    const char *printMSG = "Incorrect number of arguments: Usage: \n\
			    ./cuising filename L T N_steps period burnin \n";
    if (argc < 7){
        printf("%s", printMSG);
	return 0;
    }
    else if (argc > 7){
        printf("%s", printMSG);
        return 0;
    }

    char *filename = argv[1];
    int L = atoi(argv[2]), N_steps = atoi(argv[4]);
    float T = atof(argv[3]);
    int period = atoi(argv[5]), burnin = atoi(argv[6]);
    printf("Saving to %s with L=%d, T=%f, every %d steps,\n with burnin=%d\n",
           filename, L, T, period, burnin);

    int N = L*L;

    hiprandGenerator_t rng;
    checkCudaErrors(hiprandCreateGenerator(&rng, HIPRAND_RNG_PSEUDO_DEFAULT));
    checkCudaErrors(hiprandSetPseudoRandomGeneratorSeed(rng, 920989ULL));

    hipStream_t cpyStream, rngStream, sampleStream;
    checkCudaErrors(hipStreamCreate(&cpyStream));
    checkCudaErrors(hipStreamCreate(&rngStream));
    checkCudaErrors(hipStreamCreate(&sampleStream));
    
    checkCudaErrors(hiprandSetStream(rng, rngStream));

    int *h_spins = (int *)malloc(sizeof(int) * N);
    memset(h_spins, 1, sizeof(int) * N);

    for (int i = 0; i < N; i++){
        float r = (float)rand()/RAND_MAX;
        h_spins[i] = (r > 0.5) ? 1 : -1;
    }
    int *d_spins;
    float *d_random; 
    checkCudaErrors(hipMalloc((void **)&d_spins, sizeof(int) * N));
    checkCudaErrors(hipMalloc((void **)&d_random, sizeof(float) * N));
    checkCudaErrors(hipMemcpy(d_spins, h_spins, sizeof(int) * N, hipMemcpyHostToDevice));

    checkCudaErrors(hiprandGenerateUniform(rng, d_random, N));

#ifdef DBUG
    float *h_random = (float *)malloc(sizeof(float) * N);
    checkCudaErrors(hipMemcpy(h_random, d_random, sizeof(int) * N, hipMemcpyDeviceToHost));
    FILE *fp = fopen("dbug.dat", "w");
    for (int i=0; i < N; i++){
        if (i%L ==0)
            fprintf(fp, "\n");
        fprintf(fp, "%d\t", h_spins[i]);
    }
    for (int i=0; i < N; i++){
        if (i%L ==0)
            fprintf(fp, "\n");
        fprintf(fp, "%f\t", h_random[i]);
    }
    free(h_random);
#endif 

    int BLOCKS_X = ceil((float)L/(float)THREAD_X);
    int BLOCKS_Y = ceil((float)L/(float)THREAD_Y);
    int BLOCKMEM = sizeof(int) * (THREAD_X+2) * (THREAD_Y+2);
    dim3 blocks(BLOCKS_X, BLOCKS_Y);
    dim3 threads(THREAD_X+2, THREAD_Y+2); //include boundary

    hipEvent_t start, stop;
    float time = 0.f;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));
    checkCudaErrors(hipEventRecord(start, 0));
   

    for (int t = 0; t < burnin; t++){
        checkCudaErrors(hipStreamSynchronize(rngStream));
        isingSample<<<blocks, threads, 
                      BLOCKMEM, sampleStream>>>(d_spins, d_random, T, L);
        checkCudaErrors(hiprandGenerateUniform(rng, d_random, N));
    } 
    
    FILE *fpSave = fopen(filename, "w");

    for (int t = 0; t < N_steps; t++){
        checkCudaErrors(hipStreamSynchronize(rngStream));
        isingSample<<<blocks, threads, 
                      BLOCKMEM, sampleStream>>>(d_spins, d_random, T, L);
        checkCudaErrors(hiprandGenerateUniform(rng, d_random, N));
        //TODO: make bitpacking function
        //if (t % period == 0){
        //    checkCudaErrrors(hipMemcpyAsync(h_spins, d_spins, sizeof(int)*N,
        //                                     hipMemcpyDeviceToHost, cpyStream));
        //    for (int i=0; i < N; i++){
        //        fprintf(fpSave, "%f\t", h_spin[i]);
        //    }
        //    fprintf(fpSave, "\n");
        //}
    }

    checkCudaErrors(hipMemcpy(h_spins, d_spins, sizeof(int) * N, hipMemcpyDeviceToHost));
    for (int i=0; i < N; i++){
        if (i%L ==0)
            fprintf(fpSave, "\n");
        fprintf(fpSave, "%d\t", h_spins[i]);
    }
    fclose(fpSave);

    checkCudaErrors(hipEventRecord(stop, 0));
    checkCudaErrors(hipEventSynchronize(stop));
    hipEventElapsedTime(&time, start, stop);
    printf("Elapsed time: %f ms, %f ms / site updated\n", time, time/(float)(burnin+N_steps+N));
    
#ifdef DBUG
    for (int i=0; i < N; i++){
        if (i%L ==0)
            fprintf(fp, "\n");
        fprintf(fp, "%d\t", h_spins[i]);
    }
    fclose(fp);
#endif
   
    checkCudaErrors(hipStreamDestroy(cpyStream));
    checkCudaErrors(hipStreamDestroy(rngStream));
    checkCudaErrors(hipStreamDestroy(sampleStream));
    checkCudaErrors(hiprandDestroyGenerator(rng));

    hipFree(d_spins);
    hipFree(d_random);
    free(h_spins);
    checkCudaErrors(hipGetLastError());

    return EXIT_SUCCESS;
}


