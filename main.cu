#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <float.h>
#include <string.h>

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hiprand.h>
#include <hip/hip_runtime_api.h> //In samples/common/inc
#include <thrust/reduce.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include <sampling.h>

//#define DBUG //Save stuff to files
#define MOD(x, N) (((x < 0) ? ((x % N) + N) : x) % N)
#ifndef MIN
#define MIN(a, b) ((a > b) ? b : a)
#endif
#ifndef MAX
#define MAX(a, b) ((a > b) ? a : b)
#endif

#define THREAD_X 64
#define THREAD_Y 8

int main(int argc, char **argv){

    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    if (deviceCount == 0){
        fprintf(stderr, "Error: no CUDA supporting devices.\n");
	exit(EXIT_FAILURE);
    }
    int dev = 0; 
    hipSetDevice(dev);
    
    const char *printMSG = "Incorrect number of arguments: Usage: \n\
			    ./cuising filename L T N_steps period burnin \n";
    if (argc < 7){
        printf("%s", printMSG);
	return 0;
    }
    else if (argc > 7){
        printf("%s", printMSG);
        return 0;
    }

    char *filename = argv[1];
    int L = atoi(argv[2]), N_steps = atoi(argv[4]);
    float T = atof(argv[3]);
    int period = atoi(argv[5]), burnin = atoi(argv[6]);
    printf("Saving to %s with L=%d, T=%f, every %d steps,\n with burnin=%d\n",
           filename, L, T, period, burnin);

    int N = L*L;

    hiprandGenerator_t rng;
    checkCudaErrors(hiprandCreateGenerator(&rng, HIPRAND_RNG_PSEUDO_DEFAULT));
    checkCudaErrors(hiprandSetPseudoRandomGeneratorSeed(rng, 920989ULL));

    //hipblasHandle_t handle;
    //checkCudaErrors(hipblasCreate(&handle));

    int *h_spins = (int *)malloc(sizeof(int) * N);
    memset(h_spins, 1, sizeof(int) * N);

    for (int i = 0; i < N; i++){
        float r = (float)rand()/RAND_MAX;
        h_spins[i] = (r > 0.5) ? 1 : -1;
    }
    int *d_spins;
    float *d_random;
    checkCudaErrors(hipMalloc((void **)&d_spins, sizeof(int) * N));
    checkCudaErrors(hipMalloc((void **)&d_random, sizeof(float) * N));
    checkCudaErrors(hipMemcpy(d_spins, h_spins, sizeof(int) * N, hipMemcpyHostToDevice));
    checkCudaErrors(hipDeviceSynchronize());

    checkCudaErrors(hiprandGenerateUniform(rng, d_random, N));
    checkCudaErrors(hipDeviceSynchronize());
    float *h_random = (float *)malloc(sizeof(float) * N);
    checkCudaErrors(hipMemcpy(h_random, d_random, sizeof(int) * N, hipMemcpyDeviceToHost));
    checkCudaErrors(hipDeviceSynchronize());

    FILE *fp = fopen("dbug.dat", "w");
    for (int i=0; i < N; i++){
        if (i%L ==0)
            fprintf(fp, "\n");
        fprintf(fp, "%d\t", h_spins[i]);
    }
    for (int i=0; i < N; i++){
        if (i%L ==0)
            fprintf(fp, "\n");
        fprintf(fp, "%f\t", h_random[i]);
    }

    int BLOCKS_X = ceil((float)L/(float)THREAD_X);
    int BLOCKS_Y = ceil((float)L/(float)THREAD_Y);
    int BLOCKMEM = sizeof(int) * (THREAD_X+2) * (THREAD_Y+2);
    dim3 blocks(BLOCKS_X, BLOCKS_Y);
    dim3 threads(THREAD_X+2, THREAD_Y+2);

    hipEvent_t start, stop;
    float time = 0.f;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));
    checkCudaErrors(hipEventRecord(start, 0));
   

    for (int t = 0; t < burnin; t++){
        isingSample<<<blocks, threads, 
                      BLOCKMEM>>>(d_spins, d_random, T, L);
        checkCudaErrors(hiprandGenerateUniform(rng, d_random, N));
        //checkCudaErrors(hipDeviceSynchronize());
    } 
    
    for (int t = 0; t < N_steps; t++){
        isingSample<<<blocks, threads, 
                      BLOCKMEM>>>(d_spins, d_random, T, L);
        checkCudaErrors(hiprandGenerateUniform(rng, d_random, N));
        //checkCudaErrors(hipDeviceSynchronize());
    }

    checkCudaErrors(hipEventRecord(stop, 0));
    checkCudaErrors(hipEventSynchronize(stop));
    hipEventElapsedTime(&time, start, stop);
    printf("Elapsed time: %f ms, %f ms / site updated\n", time, time/(float)(burnin+N_steps+N));
    
    checkCudaErrors(hipMemcpy(h_spins, d_spins, sizeof(int) * N, hipMemcpyDeviceToHost));

    for (int i=0; i < N; i++){
        if (i%L ==0)
            fprintf(fp, "\n");
        fprintf(fp, "%d\t", h_spins[i]);
    }

    fclose(fp);
    hipFree(d_spins);
    hipFree(d_random);
    free(h_spins);
    free(h_random);
    checkCudaErrors(hipGetLastError());

    return EXIT_SUCCESS;
}


